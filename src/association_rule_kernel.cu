#include "hip/hip_runtime.h"
#include "association_rule.h"
__device__ bool is_sub_array(Pattern* pattern, Trans* trans) {
    
    //printf("enter sub array\n");
    // whether matches            
    bool ret = false;
    int pat_data_num = pattern->pat_num;
    int input_data_num = trans->num;
    //printf("here\n");
    if (pat_data_num <= input_data_num) {
        int input_ptr = 0;
        int pat_ptr = 0;
        while ( (pat_ptr < pat_data_num) && (input_ptr < input_data_num) ) {
            if (pattern->data[pat_ptr] < trans->data[input_ptr]) break;
            else if (pattern->data[pat_ptr] == trans->data[input_ptr]) {
                pat_ptr ++;
                input_ptr ++;
            } else {
                input_ptr ++;
            }
        }
        if (pat_ptr == pat_data_num)
            ret = true; 
    }                           
    return ret;
}

__device__ bool generate_new_pattern (Pattern* old, Pattern* cur, Pattern* tail) {

    // //bool ret = false;
    // if ( (old->pat_num == cur->pat_num) && (old->num > THREASHOLD) ) {
    //     //int m = 0;
    //     for (int m = 0; m < cur->pat_num-1; ++m) {
    //         if (old->data[m] != cur->data[m]) {
    //             return false;
    //         }
    //     }
                
        
    //     tail->pat_num = old->pat_num;
    //     tail->num = 0;
    //     for (int i = 0; i < old->pat_num; ++i) {
    //             tail->data[i] = old->data[i];
    //     }
    //         tail->data[old->pat_num] = cur->data[old->pat_num-1];
    //         return true;
        

    // }
    return false;
}

__global__ void association_kernel (Trans* input, int input_num, Pattern* pattern, int pat_num) {
    __shared__ int smem[CUDATHREAD];
    int pat_idx = 0;    
    int cmp_idx = 0;
    bool start = false;
    
   
    //one pattern a time
    while(pat_idx < pat_num) {
        //printf("%d, %d\n", pat_idx, pat_num);
        //   
        smem[threadIdx.x] = 0;
        //printf("%d, %d\n", threadIdx.x, input_num); 
        //printf("aaaa");
        // All input on this thread
        int input_idx = threadIdx.x;
        //int tmp_num = 0;
        while (input_idx < input_num) {            
            // whether matches            
            //printf("a\n");
            //if (threadIdx.x == 10) {
            //    printf("hhh\n");
            //}

            if (is_sub_array(pattern+pat_idx, input + input_idx)) {
                smem[threadIdx.x] = smem[threadIdx.x] + 1;
                //tmp_num++;
                //printf("here: %d, %d, %d\n", threadIdx.x, input_idx, input_num);    
            }
            input_idx += blockDim.x;
            //printf("here!\n");
        }
       

        //if (threadIdx.x < 512) {
        //            printf("Wrong\n");
        //}

        //printf("hhh\n");
        
        //
        __syncthreads();        
        //smem[threadIdx.x] = tmp_num;
        //__syncthreads();
        //sum up this pattern
        for (int i = (blockDim.x >> 1); i > 0; i >>=1 ) {
            if (threadIdx.x < i) {
                //printf("%d,%d\n", threadIdx.x, i);
                smem[threadIdx.x] += smem[threadIdx.x + i];
            }
            __syncthreads();
        }
        //printf("here?\n");
        bool mybool = true;
        if (threadIdx.x == 0) {
            // write back
            //printf("%d, %d\n", pat_idx, pat_num);
            pattern[pat_idx].num = smem[0];
            //printf("num: %d\n", pattern[pat_idx].num);
            smem[0] = pat_num;
            if (pattern[pat_idx].num > THREASHOLD) {
                start = false;
                for (int k = cmp_idx; k < pat_idx; k++) {
                //for (int k = 0; k < pat_idx; k++) {  
                    mybool = true;
                                      
                    if ( (pattern[k].pat_num == pattern[pat_idx].pat_num) && (pattern[k].num > THREASHOLD) ) {
    
                        for (int m = 0; m < pattern[k].pat_num-1; ++m) {
                            if (pattern[k].data[m] != pattern[pat_idx].data[m]) {
                                mybool = false;
                                break;
                            }
                        }
                        //printf("%d, %d\n", k, pat_num);
                
                        if (mybool) {
                            //printf("%d,%d,%d\n", k, pat_idx, pat_num);
                            pattern[pat_num].pat_num = pattern[k].pat_num+1;
                            pattern[pat_num].num = 0;
                            
                            for (int i = 0; i < pattern[k].pat_num; ++i) {
                                pattern[pat_num].data[i] = pattern[k].data[i];
                                //printf("%i, %d, %d\n", i, pattern[pat_idx].data[i] , pattern[k].data[i]);
                            }
                            pattern[pat_num].data[pattern[k].pat_num] = pattern[pat_idx].data[pattern[k].pat_num-1];
                            if (!start) {
                                start = true;
                                cmp_idx = k;
                            }
                            //printf("here\n");
                            ++pat_num;
                            smem[0] = pat_num;
                        } 
        
                        
                    }
                    
                }
            }  
            //printf("yy\n");
        } 
        __syncthreads();

        pat_num = smem[0];
        pat_idx ++;
        //printf("%d, %d\n", pat_idx, pat_num);

    }

    if (threadIdx.x == 0)
        printf("Finish, %d\n", pat_num);
}

