#include "hip/hip_runtime.h"
#include "association_rule.h"
__device__ bool is_sub_array(Trans* trans, Pattern* pattern) {
         
    // whether matches            
    bool ret = false;
    int pat_data_num = pattern.pat_num;
    int input_data_num = trans.num;
    if (pat_data_num <= input_data_num) {
        int input_ptr = 0;
        int pat_ptr = 0;
        while ( (pat_ptr < pat_data_num) && (input_ptr < input_data_num) ) {
            if (pattern.data[pat_ptr] < trans.data[input_ptr]) break;
            else if (pattern.data[pat_ptr] == trans.data[input_ptr]) {
                pat_ptr ++;
                input_ptr ++;
            } else {
                input_ptr ++;
            }
        }
        if (pat_ptr == pat_data_num)
            ret = true; 
    }                           
    return ret;
}

__device__ bool generate_new_pattern (Pattern* old, Pattern* cur, Pattern* tail) {

    bool ret = false;
    if ( (old->pat_num == cur->pat_num) && (old->num > THREASHOLD) ) {
        int m = 0;
        for (; m < cur->pat_num-1; ++m) {
            if (old->data[m] != cur->data[m]) {
                break;
            }
        }
                
        if (m == cur->pat_num-1) {
            tail.pat_num = old.pat_num;
            tail.num = 0;
            for (int i = 0; i < old.pat_num; ++i) {
                tail.data[m] = tail.data[m];
            }
            tail.data[pat.pat_num] = cur.data[pat.pat_num-1];
            ret = true;
        }

    }
    return ret;
}

__global__ void association_kernel (Trans* input, int input_num, Pattern* pattern, int pat_num, int* ret) {
    
    int pat_idx = 0;    
    int cmp_idx = 0;

    extern __shared__ int smem[];

    //one pattern a time
    while(pat_idx < pat_num) {
        smem[threadIdx.x] = 0;
        // All input on this thread
        int input_idx = threadIdx.x;
        while (input_idx < input_num) {            
            // whether matches            
            if (is_sub_array(pattern+pat_idx, input + input_idx)) {
                smem[threadIdx.x] ++;
            }
            input_idx += blockDim.x;
        }
        __syncthreads();        
        //sum up this pattern
        for (int i = (blockDim.x >> 1); i > 0; i >>=1 ) {
            if (threadIdx.x < i) {
                smem[threadIdx.x] += smem[threadIdx.x + i];
            }
            __syncthreads();
        }


        if (threadIdx.x == 0) {
            // write back
            pattern[pat_idx].num = smem[0];
            smem[0] = pat_num;
            int k = cmp_idx;
            if (pattern[pat_idx].num > THREASHOLD) {
                bool start = false;
                for (; k < pat_idx; k++) {
                    if (generate_new_pattern(pattern+k, pattern+pat_idx, pattern+pat_num))
                        if (!start) {
                            start = true;
                            cmp_idx = k;   
                        }
                        smem[0] = pat_num++;
                    }
                }
            }  
        } 
        __syncthreads();

        pat_num = smem[0];
        pat_idx ++;


    }
    *ret = pat_num;
}

