#include "association_rule.h"
Trans trans[TRANSNUM];
int tnum = 0;
int sdata[ITEMSIZE]; 
//int sdata_num = 0;

void apply_association(std::map<int, int>&itemmp){
    for (int i = 0; i < tnum; i ++) {
        for (int j = 0; j < trans[i].num; ++j) {
            int data = trans[i].data[j];
            if (data == 1) {
            }
            if (itemmp.find(data) == itemmp.end()) {
                itemmp[data] = 1;
            } else {
                itemmp[data] ++;
            }
            //cout << "a" << endl;
        }
    }
    return;
}

int main() {

    ifstream fin;
    fin.open("BMS1_itemset_mining.txt");
    string line;
    tnum = 0;
    while(getline(fin, line)) {
        //cout << "zz" << std::endl;
        stringstream ss(line);
        int x;
        int dnum = 0;
        while(ss >> x) {
            trans[tnum].data[dnum++] = x;
            
        }
        if ((dnum <= DATA_SIZE)) {
            trans[tnum++].num = dnum;
        }  
        if (tnum > 50000) {
            break;
        }
    }
    std::map<int, int> itemmp;
    apply_association(itemmp);
    
        
    
    //int pat_size = 0;
    int sdata_num = 0;
    for (std::map<int,int>::iterator iter = itemmp.begin(); iter != itemmp.end(); ++iter) {
        
        if (iter->second > THREASHOLD) {
            sdata[sdata_num++] = iter->first;
        }
    }
    sort(sdata, sdata+sdata_num);
    
    //for (int i = 0; i < sdata_num; ++i) {
    //    cout << sdata[i] <<'\t';
    //}
    cout << tnum << endl;
    cout << sdata_num << endl;
    int dual_size = (sdata_num-1)*sdata_num/2;
    Pattern* pattern = new Pattern[dual_size];
    
    
    int dual_ptr = 0;
    int per_size = dual_size/STREAM_NUM;
    int pos = 0;
    int pos_array[STREAM_NUM];
    int size_array[STREAM_NUM];
    pos_array[0] = 0;
    for (int  i = 0; i < sdata_num-1; ++i) {
        for (int j = i+1; j < sdata_num; ++j) {
            pattern[dual_ptr].pat_num = 2;
            pattern[dual_ptr].num = 0;
            pattern[dual_ptr].data[0] = sdata[i];
            pattern[dual_ptr].data[1] = sdata[j];
            dual_ptr++;
        }
        if (pos < STREAM_NUM - 1) {
            if (per_size < dual_ptr - pos_array[pos]) {
                size_array[pos] = dual_ptr - pos_array[pos];   
                pos ++;
                if (pos < STREAM_NUM) {
                    pos_array[pos] = dual_ptr;
                } 
            }
        }
    }
    size_array[pos] = dual_ptr - pos_array[pos];
    

    for (int i = 0; i < STREAM_NUM; ++i) {
        cout << pos_array[i]<<" " << size_array[i] << endl;
    }
    
    
    hipStream_t streams[STREAM_NUM];
    Pattern *device_pattern[STREAM_NUM];
    Trans* device_trans;
    
    int *device_pat_num;
    int *host_pat_num;
    
    for (int i = 0; i < STREAM_NUM; ++i) {
        hipStreamCreate(&streams[i]);
        hipMalloc((void**)&device_pattern[i], 300000*sizeof(Pattern));
    }

    hipMalloc((void**)&device_trans, tnum * sizeof(Trans));

    hipMalloc((void**)&device_pat_num, STREAM_NUM*sizeof(int));
    hipMemcpy(device_trans,  trans, tnum*sizeof(Trans), hipMemcpyHostToDevice);
    for (int i = 0; i < STREAM_NUM; ++i) {
        hipMemcpyAsync(device_pattern[i], &pattern[pos_array[i]], size_array[i] * sizeof(Pattern), hipMemcpyHostToDevice, streams[i]);
        int threadnum = 512;
        dim3 threadDim(threadnum);
        dim3 blockDim(1);
        association_kernel<<<blockDim, threadDim, 0 ,streams[i]>>>(device_trans,tnum, device_pattern[i], size_array[i]); 
    }


    for (int i = 0; i < STREAM_NUM; ++i) {
        Pattern* output_pattern = new Pattern[PERMEM/sizeof(Pattern)];
        hipMemcpyAsync(output_pattern, device_pattern[i], PERMEM, hipMemcpyDeviceToHost, streams[i]);
    }
    host_pat_num = new int[STREAM_NUM];
    hipMemcpy(host_pat_num, device_pat_num, STREAM_NUM*sizeof(int), hipMemcpyDeviceToHost);


    return 0;
}