#include "association_rule.h"
#include <sys/time.h>
Trans trans[TRANSNUM];
int tnum = 0;
int sdata[ITEMSIZE]; 
//int sdata_num = 0;

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void apply_association(std::map<int, int>&itemmp){
    for (int i = 0; i < tnum; i ++) {
        for (int j = 0; j < trans[i].num; ++j) {
            int data = trans[i].data[j];
            if (data == 1) {
            }
            if (itemmp.find(data) == itemmp.end()) {
                itemmp[data] = 1;
            } else {
                itemmp[data] ++;
            }
            //cout << "a" << endl;
        }
    }
    return;
}

int main() {

    ifstream fin;
    fin.open("mushrooms.txt");
    string line;
    tnum = 0;
    while(getline(fin, line)) {
        //cout << "zz" << std::endl;
        stringstream ss(line);
        int x;
        int dnum = 0;
        while(ss >> x) {
            trans[tnum].data[dnum++] = x;
            
        }
        if ((dnum <= DATA_SIZE)) {
            trans[tnum++].num = dnum;
        }  
        if (tnum > 50000) {
            break;
        }
    }
    double prestart = cpuSecond();
    std::map<int, int> itemmp;
    apply_association(itemmp);
    
        
    
    //int pat_size = 0;
    int sdata_num = 0;
    for (std::map<int,int>::iterator iter = itemmp.begin(); iter != itemmp.end(); ++iter) {
        
        if (iter->second > THREASHOLD) {
            sdata[sdata_num++] = iter->first;
        }
    }
    sort(sdata, sdata+sdata_num);
    
    //for (int i = 0; i < sdata_num; ++i) {
    //    cout << sdata[i] <<'\t';
    //}
    cout << tnum << endl;
    cout << sdata_num << endl;
    int dual_size = (sdata_num-1)*sdata_num/2;
    Pattern* pattern = new Pattern[dual_size];
    
    
    int dual_ptr = 0;
    int per_size = dual_size/STREAM_NUM;
    int pos = 0;
    int pos_array[STREAM_NUM];
    int size_array[STREAM_NUM];
    pos_array[0] = 0;
    for (int  i = 0; i < sdata_num-1; ++i) {
        for (int j = i+1; j < sdata_num; ++j) {
            pattern[dual_ptr].pat_num = 2;
            pattern[dual_ptr].num = 0;
            pattern[dual_ptr].data[0] = sdata[i];
            pattern[dual_ptr].data[1] = sdata[j];
            dual_ptr++;
        }
        if (pos < STREAM_NUM - 1) {
            if (per_size < dual_ptr - pos_array[pos]) {
                size_array[pos] = dual_ptr - pos_array[pos];   
                pos ++;
                if (pos < STREAM_NUM) {
                    pos_array[pos] = dual_ptr;
                } 
            }
        }
    }
    size_array[pos] = dual_ptr - pos_array[pos];
    
    double preend = cpuSecond();
    printf("pre time: %lf\n", preend-prestart);
    for (int i = 0; i < STREAM_NUM; ++i) {
        cout << pos_array[i]<<" " << size_array[i] << endl;
    }
    
    
    hipStream_t streams[STREAM_NUM];
    Pattern *device_pattern[STREAM_NUM];
    Trans* device_trans;
    
    int *device_pat_num;
    int *host_pat_num;
    
    for (int i = 0; i < STREAM_NUM; ++i) {
        hipStreamCreate(&streams[i]);
        hipMalloc((void**)&device_pattern[i], PATTERNNUM*sizeof(Pattern));
    }

    hipMalloc((void**)&device_trans, tnum * sizeof(Trans));

    hipMalloc((void**)&device_pat_num, STREAM_NUM*sizeof(int));
    double start_time = cpuSecond();
    hipMemcpy(device_trans,  trans, tnum*sizeof(Trans), hipMemcpyHostToDevice);
    for (int i = 0; i < STREAM_NUM; ++i) {
        hipMemcpyAsync(device_pattern[i], &pattern[pos_array[i]], size_array[i] * sizeof(Pattern), hipMemcpyHostToDevice, streams[i]);
        int threadnum = CUDATHREAD;
        dim3 threadDim(threadnum);
        dim3 blockDim(1);
        association_kernel<<<blockDim, threadDim, 0 ,streams[i]>>>(device_trans,tnum, device_pattern[i], size_array[i]); 
    }


    for (int i = 0; i < STREAM_NUM; ++i) {
        Pattern* output_pattern = new Pattern[PATTERNNUM];
        hipMemcpyAsync(output_pattern, device_pattern[i], PATTERNNUM*sizeof(Pattern), hipMemcpyDeviceToHost, streams[i]);
    }
    //double end_time = cpuSecond();
    //printf("time: %lf\n", end_time-start_time);
    //printf("size of pattern:%d\n", sizeof(Pattern));
    //host_pat_num = new int[STREAM_NUM];
    //hipMemcpy(host_pat_num, device_pat_num, STREAM_NUM*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    double end_time = cpuSecond();
    printf("time: %lf\n", end_time-start_time);

    return 0;
}