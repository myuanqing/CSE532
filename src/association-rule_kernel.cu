#include "hip/hip_runtime.h"
#define DATA_SIZE 30
#define THREASHOLD 10
struct Trans {
    int num;
    int data[DATA_SIZE];
};

struct Pattern {
    int num;
    int pat_num;
    int data[DATA_SIZE];
};


__global__ void* association_kernel (Trans* input, int input_num, Pattern* pattern, int* pat_data_array, int pat_num, int pattern_dim) {
    
    pattern += pattern_dim * blockIdx.x;
    //int tid = threadIdx.x + blockIdx.x * blockDim.x;    
    int pat_idx = 0;
    
    int cmp_idx = 0;
    extern __shared__ int smem[];

    //one pattern a time
    while(pat_idx < pat_num) {
        smem[threadIdx.x] = 0;
        Pattern pat = pattern[pat_idx];
        // All input on this thread
        int input_idx = threadIdx.x;
        while (input_idx < input_num) {
            Trans ipt = input[input_idx];            
            // whether matches            
            int pat_data_num = pat.pat_num;
            int input_data_num = ipt.num;
            if (pat_data_num <= input_data_num) {
                int input_ptr = 0;
                int pat_ptr = 0;
                while ( (pat_ptr < pat_data_num) && (input_ptr < input_data_num) ) {
                    if (pat.data[pat_ptr] < ipt.data[input_ptr]) break;
                    else if (pat.data[pat_ptr] == ipt.data[input_ptr]) {
                        pat_ptr ++;
                        input_ptr ++;
                    } else {
                        input_ptr ++;
                    }
                }
                if (pat_ptr == pat_data_num)
                    smem[threadIdx.x] ++; 
            }                           
            input_idx += blockDim.x;

        }
        __syncthreads();
        //sum up this pattern
        for (int i = (blockDim.x >> 1); i > 0; i >>=1 ) {
            if (threadIdx.x < i) {
                smem[threadIdx.x] += smem[threadIdx.x + i];
            }
            __syncthreads();
        }


        if (threadIdx.x == 0) {
            // write back
            pat.num = smem[0];
            smem[0] = pat_num;
            int k = cmp_idx;
            if (pat.num > THREASHOLD) {
                bool start = false;
                for (; k < pat_idx; k++) {
                    if ( (pattern[k].pat_num == pat.pat_num) && (pattern[k].num > THREASHOLD) ) {
                        bool test = true;
                        for (int m = 0; m < pat.pat_num-1; ++m) {
                            if (pattern[k].data[m] != pat.data[m]) {
                                test = false;
                                break;
                            }
                        }
                        if (test && (!start)) {
                            start = true;
                            cmp_idx = k;   
                        }
                        if (test) {
                            pattern[pat_num].pat_num= pat.pat_num;
                            pattern[pat_num].num = 0;
                            for (int m = 0; m < pat.pat_num; ++m) {
                                pattern[pat_num].data[m] = pattern[k].data[m];
                            }
                            pattern[pat_num].data[pat.pat_num] = pat.data[pat.pat_num-1];
                            smem[0] = pat_num++;

                        }

                    }
                }
            }  
        } 
        __syncthreads();

        pat_num = smem[0];
        pat_idx ++;


    }
}